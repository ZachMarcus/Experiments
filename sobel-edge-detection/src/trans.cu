
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        exit(-1);
    }
    #endif
    return result;
}
                
// GPU kernel and functions
__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int height,
                       unsigned int width){
        
    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    
       // http://homepages.inf.ed.ac.uk/rbf/HIPR2/sobel.htm
       // https://stackoverflow.com/questions/14358916/applying-sobel-edge-detection-with-cuda-and-opencv-on-a-grayscale-jpg-image 
    if (x < width && y < height ){
        int gX[3][3] = {{-1,0,1},{-2,0,2},{-1,0,1}};
        int gY[3][3] = {{-1,-2,-1},{0,0,0},{1,2,1}};

        double x_sum = 0.0;
        double y_sum = 0.0;
        int index;
        for (int j = -1; j < 2; j++) {
            for (int i = -1; i < 2; i++) {
                index = width * (j + y) + i + x;
                x_sum += input[index] * gX[j+1][i+1];
                y_sum += input[index] * gY[j+1][i+1];
            }
        }

        double answer = sqrt(x_sum * x_sum + y_sum * y_sum);
        // keep exceeding values, so correct that, and apply filter
        if (answer < 128) {answer = 0;} else {
            answer = 255;
        }
      
        output[x*height+y] = answer;
    }
}

void transpose_img(unsigned char *in_mat, 
                   unsigned char *out_mat, 
                   unsigned int height, 
                   unsigned int width){
                         
    int gridXSize = 1 + (( width - 1) / TILE_SIZE);
    int gridYSize = 1 + ((height - 1) / TILE_SIZE);
    
    int XSize = gridXSize*TILE_SIZE;
    int YSize = gridYSize*TILE_SIZE;
    
    // Both are the same size (CPU/GPU).
    int size = XSize*YSize;
    
    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&input_gpu    , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
    
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
                        in_mat, 
                        height*width*sizeof(char), 
                        hipMemcpyHostToDevice));

    checkCuda(hipDeviceSynchronize());
    
    // Execute algorithm
    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    #if defined(CUDA_TIMING)
        float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif
    
    // Kernel Call
    kernel<<<dimGrid, dimBlock>>>(input_gpu, output_gpu, height, width);
    
    checkCuda(hipDeviceSynchronize());
    
    #if defined(CUDA_TIMING)
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif
        
    // Retrieve results from the GPU
    checkCuda(hipMemcpy(out_mat, 
                        output_gpu, 
                        height*width*sizeof(unsigned char), 
                        hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
    checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));

}
