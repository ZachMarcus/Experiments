
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned int *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        exit(-1);
    }
    #endif
    return result;
}
                
// GPU kernel and functions
__global__ void kernel(unsigned char *input, 
                       unsigned int *output,
                       unsigned int numberOfPixels){
//    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
//    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixel < numberOfPixels) {
        atomicAdd(&(output[input[pixel]]), 1);
    }
}


void calculateHistogram(unsigned char *inputImageData,
                          unsigned int height,
                          unsigned int width,
                          unsigned int* outputHistogram) {
    unsigned int numberOfPixels = height * width;
    int gridXSize = 1 + (( width - 1) / TILE_SIZE);
    int gridYSize = 1 + ((height - 1) / TILE_SIZE);

    int XSize = gridXSize*TILE_SIZE;
    int YSize = gridYSize*TILE_SIZE;

    // Both are the same size (CPU/GPU).
    int size = XSize*YSize;

    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&input_gpu    , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned int)));
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned int)));
    checkCuda(hipMemcpy(input_gpu, inputImageData, height*width*sizeof(char),hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
    
    // Execute algorithm
    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    #if defined(CUDA_TIMING)
        float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif

    // Kernel Call
    kernel<<<dimGrid, dimBlock>>>(input_gpu, output_gpu, numberOfPixels);
    checkCuda(hipDeviceSynchronize());
    #if defined(CUDA_TIMING)
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif
        
    // Retrieve results from the GPU
    checkCuda(hipMemcpy(outputHistogram, 
                        output_gpu, 
                        256 * sizeof(unsigned int), 
                        hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
    checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));

}
